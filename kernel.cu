#include "hip/hip_runtime.h"
#include <stdio.h>
#include <float.h>
#include <inttypes.h>
#include "kmcuda.h"

#define BLOCK_SIZE 1024

__device__ float dist_sum;
__device__ uint32_t changed;
__constant__ uint32_t samples_size;
__constant__ uint16_t features_size;
__constant__ uint32_t clusters_size;

__global__ void kmeans_plus_plus(uint32_t cc, float *samples,
                                 float *centroids, float *dists) {
  uint32_t sample = blockIdx.x * blockDim.x + threadIdx.x;
  if (sample >= samples_size) {
    return;
  }
  uint32_t soffset = sample * features_size;
  extern __shared__ float local_dists[];
  float min_dist = FLT_MAX;
  for (uint32_t c = 0; c < cc; c++) {
    float dist = 0;
    uint32_t coffset = c * features_size;
    for (uint16_t f = 0; f < features_size; f++) {
      float myf = samples[soffset + f];
      float d = myf - centroids[coffset + f];
      dist += d * d;
    }
    if (dist < min_dist) {
      min_dist = dist;
    }
  }
  min_dist = sqrt(min_dist);
  dists[sample] = min_dist;
  local_dists[threadIdx.x] = min_dist;
  __syncthreads();
  if (threadIdx.x == 0) {
    float local_sum = 0;
    for (int i = 0; i < blockDim.x; i++) {
      local_sum += local_dists[i];
    }
    dist_sum += local_sum;
  }
}

__global__ void kmeans_assign(float *samples, float *centroids,
                              uint32_t *ccounts, uint32_t *assignments) {
  uint32_t sample = blockIdx.x * blockDim.x + threadIdx.x;
  if (sample >= samples_size) {
    return;
  }
  uint32_t soffset = sample * features_size;
  float min_dist = FLT_MAX;
  uint32_t nearest = UINT32_MAX;
  for (uint32_t c = 0; c < clusters_size; c++) {
    float dist = 0;
    uint32_t coffset = c * features_size;
    for (int f = 0; f < features_size; f++) {
      float myf = samples[soffset + f];
      float d = myf - centroids[coffset + f];
      dist += d * d;
    }
    if (dist < min_dist) {
      min_dist = dist;
      nearest = c;
    }
  }
  if (nearest == UINT32_MAX) {
    printf("CUDA kernel kmeans_assign: nearest neighbor search failed for"
           "sample %" PRIu32, samples);
    return;
  }
  if (assignments[sample] != nearest) {
    assignments[sample] = nearest;
    atomicAdd(&changed, 1);
  }
  atomicAdd(&ccounts[nearest], 1);
}

__global__ void kmeans_sum(float *samples, float *centroids,
                           uint32_t *assignments) {
  uint32_t sample = blockIdx.x * blockDim.x + threadIdx.x;
  if (sample >= samples_size) {
    return;
  }
  uint32_t soffset = sample * features_size;
  uint32_t coffset = assignments[sample] * features_size;
  for (int f = 0; f < features_size; f++) {
    float myf = samples[soffset + f];
    centroids[coffset + f] += myf;
  }
}

__global__ void kmeans_adjust(float *centroids, uint32_t *ccounts) {
  uint32_t c = blockIdx.x * blockDim.x + threadIdx.x;
  if (c >= clusters_size) {
    return;
  }
  uint32_t coffset = c * features_size;
  for (int f = 0; f < features_size; f++) {
    centroids[coffset + f] /= ccounts[c];
  }
  ccounts[c] = 0;
}

extern "C" {

KMCUDAResult kmeans_cuda_setup(uint32_t samples_size_, uint16_t features_size_,
                               uint32_t clusters_size_) {
  if (hipMemcpyToSymbol(HIP_SYMBOL(samples_size), &samples_size_, sizeof(samples_size))
      != hipSuccess) {
    return kmcudaMemoryCopyError;
  }
  if (hipMemcpyToSymbol(HIP_SYMBOL(features_size), &features_size_, sizeof(features_size))
      != hipSuccess) {
    return kmcudaMemoryCopyError;
  }
  if (hipMemcpyToSymbol(HIP_SYMBOL(clusters_size), &clusters_size_, sizeof(clusters_size))
      != hipSuccess) {
    return kmcudaMemoryCopyError;
  }
  return kmcudaSuccess;
}

KMCUDAResult kmeans_cuda_plus_plus(
    uint32_t samples_size, uint32_t cc, float *samples, float *centroids,
    float *dists, float *dist_sum_) {
  float zero = 0;
  if (hipMemcpyToSymbol(HIP_SYMBOL(dist_sum), &zero, sizeof(zero)) != hipSuccess) {
    return kmcudaMemoryCopyError;
  }
  dim3 block(BLOCK_SIZE, 1, 1);
  dim3 grid(samples_size / block.x + 1, 1, 1);
  kmeans_plus_plus<<<grid, block, block.x * sizeof(float)>>>(
      cc, samples, centroids, dists);
  hipDeviceSynchronize();
  if (hipMemcpyFromSymbol(dist_sum_, HIP_SYMBOL(dist_sum), sizeof(float))
      != hipSuccess) {
    return kmcudaMemoryCopyError;
  }
  return kmcudaSuccess;
}

KMCUDAResult kmeans_cuda_internal(
    uint32_t samples_size, uint32_t clusters_size, int32_t verbosity,
    float *samples, float *centroids, uint32_t *ccounts, uint32_t *assignments) {
  dim3 sblock(BLOCK_SIZE, 1, 1);
  dim3 sgrid(samples_size / sblock.x + 1, 1, 1);
  dim3 cblock(BLOCK_SIZE, 1, 1);
  dim3 cgrid(clusters_size / cblock.x + 1, 1, 1);
  for (int i = 1; ; i++) {
    kmeans_assign<<<sgrid, sblock>>>(samples, centroids, ccounts, assignments);
    uint32_t changed_ = 0;
    if (hipMemcpyFromSymbol(&changed_, HIP_SYMBOL(changed), sizeof(changed_))
        != hipSuccess) {
      return kmcudaMemoryCopyError;
    }
    if (verbosity > 0) {
      printf("iteration %d: %" PRIu32 " reassignments\n", i, changed_);
    }
    if (!changed_) {
      break;
    }
    changed_ = 0;
    if (hipMemcpyToSymbol(HIP_SYMBOL(changed), &changed_, sizeof(changed_))
        != hipSuccess) {
      return kmcudaMemoryCopyError;
    }
    kmeans_sum<<<sblock, sgrid>>>(samples, centroids, assignments);
    kmeans_adjust<<<cblock, cgrid>>>(centroids, ccounts);
  }
  return kmcudaSuccess;
}
}
