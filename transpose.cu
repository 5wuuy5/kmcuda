#include "hip/hip_runtime.h"
#include "private.h"

#define TILE_DIM 32
#define BLOCK_ROWS 8

__global__ void copy_sample_t(
    uint32_t index, uint32_t samples_size, uint16_t features_size,
    const float *__restrict__ samples, float *__restrict__ dest) {
  uint32_t ti = blockIdx.x * blockDim.x + threadIdx.x;
  if (ti >= features_size) {
    return;
  }
  dest[ti] = samples[static_cast<uint64_t>(samples_size) * static_cast<uint64_t>(ti) + index];
}

__global__ void transpose(
    const float *__restrict__ input, uint32_t rows, uint32_t cols,
    float *__restrict__ output) {
  __shared__ float tile[TILE_DIM][TILE_DIM + 1];
  volatile uint64_t x = blockIdx.x * TILE_DIM + threadIdx.x;
  volatile uint64_t y = blockIdx.y * TILE_DIM + threadIdx.y;

  if (x < cols && y < rows) {
    for (uint32_t j = 0;
         j < min(static_cast<unsigned long>(TILE_DIM), rows - y);
         j += BLOCK_ROWS) {
      tile[threadIdx.y + j][threadIdx.x] = input[
          static_cast<uint64_t>(y + j) * cols + x];
    }
  }

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  if (x < rows && y < cols) {
    for (uint32_t j = 0;
         j < min(static_cast<unsigned long>(TILE_DIM), cols - y);
         j += BLOCK_ROWS) {
      output[static_cast<uint64_t>(y + j) * rows + x] =
          tile[threadIdx.x][threadIdx.y + j];
    }
  }
}

extern "C" {

KMCUDAResult cuda_copy_sample_t(
    uint32_t index, uint32_t offset, uint32_t samples_size, uint16_t features_size,
    const std::vector<int> &devs, int verbosity, const udevptrs<float> &samples,
    udevptrs<float> *dest) {
  FOR_EACH_DEVI(
    dim3 block(min(1024, features_size), 1, 1);
    dim3 grid(upper(static_cast<unsigned>(features_size), block.x), 1, 1);
    copy_sample_t<<<grid, block>>>(
        index, samples_size, features_size, samples[devi].get(),
        (*dest)[devi].get() + offset);
  );
  return kmcudaSuccess;
}

KMCUDAResult cuda_transpose(
    uint32_t samples_size, uint16_t features_size, bool forward,
    const std::vector<int> &devs, int verbosity, udevptrs<float> *samples) {
  INFO("transposing the samples...\n");
  uint64_t size = static_cast<uint64_t>(samples_size) * features_size * sizeof(float);
  float *ptr;
  CUCH(hipMallocManaged(&ptr, size), kmcudaMemoryAllocationFailure);
  unique_devptr<float> managed(ptr);
  hipSetDevice(devs[0]);
  CUCH(hipMemcpy(ptr, (*samples)[0].get(), size, hipMemcpyDefault), kmcudaMemoryCopyError);
  uint32_t cols, rows;
  if (forward) {
    cols = features_size;
    rows = samples_size;
  } else {
    cols = samples_size;
    rows = features_size;
  }
  dim3 block(TILE_DIM, BLOCK_ROWS, 1);
  dim3 grid(upper(cols, static_cast<uint32_t>(TILE_DIM)), upper(rows, static_cast<uint32_t>(TILE_DIM)), 1);
  DEBUG("transpose <<<(%d, %d), (%d, %d)>>> %" PRIu32 ", %" PRIu32 "\n",
        grid.x, grid.y, block.x, block.y, rows, cols);
  FOR_EACH_DEVI(
    transpose<<<grid, block>>>(ptr, rows, cols, (*samples)[devi].get());
  );
  SYNC_ALL_DEVS;
  return kmcudaSuccess;
}

}  // extern "C"