#include "hip/hip_runtime.h"
#include <stdio.h>
#include <float.h>
#include <inttypes.h>
#include <hip/hip_vector_types.h>
#include <bits/unique_ptr.h>
#include "kmcuda.h"

#define BLOCK_SIZE 1024

__device__ uint32_t changed;
__constant__ uint32_t samples_size;
__constant__ uint16_t features_size;
__constant__ uint32_t clusters_size;
__constant__ int shmem_size;

__global__ void kmeans_plus_plus(
    uint32_t cc, float *samples, float *centroids, float *dists,
    float *dist_sums) {
  uint32_t sample = blockIdx.x * blockDim.x + threadIdx.x;
  if (sample >= samples_size) {
    return;
  }
  uint32_t soffset = sample * features_size;
  extern __shared__ float local_dists[];
  float dist = 0;
  uint32_t coffset = (cc - 1) * features_size;
  for (uint16_t f = 0; f < features_size; f++) {
    float myf = samples[soffset + f];
    float d = myf - centroids[coffset + f];
    dist += d * d;
  }
  dist = sqrt(dist);
  float prev_dist = dists[sample];
  if (dist < prev_dist || cc == 1) {
    dists[sample] = dist;
  } else {
    dist = prev_dist;
  }
  local_dists[threadIdx.x] = dist;
  __syncthreads();
  if (threadIdx.x == 0) {
    uint32_t end = blockDim.x;
    if ((blockIdx.x + 1) * blockDim.x > samples_size) {
      end = samples_size - blockIdx.x * blockDim.x;
    }
    float block_sum = 0;
    for (uint32_t i = 0; i < end; i++) {
      block_sum += local_dists[i];
    }
    dist_sums[blockIdx.x] = block_sum;
  }
}

__global__ void kmeans_assign(float *samples, float *centroids,
                              uint32_t *assignments_prev,
                              uint32_t *assignments) {
  uint32_t sample = blockIdx.x * blockDim.x + threadIdx.x;
  if (sample >= samples_size) {
    return;
  }
  uint32_t soffset = sample * features_size;
  float min_dist = FLT_MAX;
  uint32_t nearest = UINT32_MAX;
  for (uint32_t c = 0; c < clusters_size; c++) {
    float dist = 0;
    uint32_t coffset = c * features_size;
    for (int f = 0; f < features_size; f++) {
      float myf = samples[soffset + f];
      float d = myf - centroids[coffset + f];
      dist += d * d;
    }
    if (dist < min_dist) {
      min_dist = dist;
      nearest = c;
    }
  }
  if (nearest == UINT32_MAX) {
    printf("CUDA kernel kmeans_assign: nearest neighbor search failed for"
           "sample %" PRIu32, samples);
    return;
  }
  uint32_t ass = assignments[sample];
  assignments_prev[sample] = ass;
  if (ass != nearest) {
    assignments[sample] = nearest;
    atomicAdd(&changed, 1);
  }
}

__global__ void kmeans_adjust(
    float *samples, float *centroids, uint32_t *assignments_prev,
    uint32_t *assignments, uint32_t *ccounts) {
  uint32_t c = blockIdx.x * blockDim.x + threadIdx.x;
  if (c >= clusters_size) {
    return;
  }
  uint32_t coffset = c * features_size;
  uint32_t my_count = ccounts[c];
  for (int f = 0; f < features_size; f++) {
    centroids[coffset + f] *= my_count;
  }
  extern __shared__ uint32_t ass[];
  int step = shmem_size / 2;
  for (uint32_t sbase = 0; sbase < samples_size; sbase += step) {
    __syncthreads();
    if (threadIdx.x == 0) {
      int pos = sbase;
      for (int i = 0; i < step && sbase + i < samples_size; i++) {
        ass[2 * i] = assignments[pos + i];
        ass[2 * i + 1] = assignments_prev[pos + i];
      }
    }
    __syncthreads();
    for (int i = 0; i < step && sbase + i < samples_size; i++) {
      uint32_t this_ass = ass[2 * i];
      uint32_t  prev_ass = ass[2 * i + 1];
      float sign = 0;
      if (prev_ass == c && this_ass != c) {
        sign = -1;
        my_count--;
      } else if (prev_ass != c && this_ass == c) {
        sign = 1;
        my_count++;
      }
      if (sign != 0) {
        uint32_t soffset = (sbase + i) * features_size;
        for (int f = 0; f < features_size; f++) {
          centroids[coffset + f] += samples[soffset + f] * sign;
        }
      }
    }
  }
  for (int f = 0; f < features_size; f++) {
    centroids[coffset + f] /= my_count;
  }
  ccounts[c] = my_count;
}

extern "C" {

KMCUDAResult kmeans_cuda_setup(uint32_t samples_size_, uint16_t features_size_,
                               uint32_t clusters_size_, uint32_t device,
                               int32_t verbosity) {
  if (hipMemcpyToSymbol(HIP_SYMBOL(samples_size), &samples_size_, sizeof(samples_size))
      != hipSuccess) {
    return kmcudaMemoryCopyError;
  }
  if (hipMemcpyToSymbol(HIP_SYMBOL(features_size), &features_size_, sizeof(features_size))
      != hipSuccess) {
    return kmcudaMemoryCopyError;
  }
  if (hipMemcpyToSymbol(HIP_SYMBOL(clusters_size), &clusters_size_, sizeof(clusters_size))
      != hipSuccess) {
    return kmcudaMemoryCopyError;
  }
  hipDeviceProp_t props;
  if (hipGetDeviceProperties(&props, device) != hipSuccess) {
    return kmcudaRuntimeError;
  }
  int my_shmem_size = static_cast<int>(props.sharedMemPerBlock);
  if (verbosity > 1) {
    printf("GPU #%" PRIu32 " has %d bytes of shared memory per block\n",
           device, my_shmem_size);
  }
  my_shmem_size /= sizeof(uint32_t);
  if (hipMemcpyToSymbol(HIP_SYMBOL(shmem_size), &my_shmem_size, sizeof(my_shmem_size))
      != hipSuccess) {
    return kmcudaMemoryCopyError;
  }
  return kmcudaSuccess;
}

KMCUDAResult kmeans_cuda_plus_plus(
    uint32_t samples_size, uint32_t cc, float *samples, float *centroids,
    float *dists, float *dist_sum, float **dev_sums) {
  dim3 block(BLOCK_SIZE, 1, 1);
  dim3 grid(samples_size / block.x + 1, 1, 1);
  if (*dev_sums == NULL) {
    if (hipMalloc(reinterpret_cast<void**>(dev_sums),
                   grid.x * sizeof(float)) != hipSuccess) {
      return kmcudaMemoryAllocationFailure;
    }
  } else {
    if (hipMemset(*dev_sums, 0, grid.x * sizeof(float)) != hipSuccess) {
      return kmcudaRuntimeError;
    }
  }
  kmeans_plus_plus<<<grid, block, block.x * sizeof(float)>>>(
      cc, samples, centroids, dists, *dev_sums);
  std::unique_ptr<float[]> host_dist_sums(new float[grid.x]);
  if (hipMemcpy(host_dist_sums.get(), *dev_sums, grid.x * sizeof(float),
                 hipMemcpyDeviceToHost) != hipSuccess) {
    return kmcudaMemoryCopyError;
  }
  float ds = 0;
  #pragma omp simd reduction(+:ds)
  for (uint32_t i = 0; i < grid.x; i++) {
    ds += host_dist_sums[i];
  }
  *dist_sum = ds;
  return kmcudaSuccess;
}

KMCUDAResult kmeans_cuda_internal(
    float tolerance, uint32_t samples_size, uint32_t clusters_size,
    uint16_t features_size, int32_t verbosity,
    float *samples, float *centroids, uint32_t *ccounts,
    uint32_t *assignments_prev, uint32_t *assignments) {
  dim3 sblock(BLOCK_SIZE, 1, 1);
  dim3 sgrid(samples_size / sblock.x + 1, 1, 1);
  dim3 cblock(BLOCK_SIZE, 1, 1);
  dim3 cgrid(clusters_size / cblock.x + 1, 1, 1);
  uint32_t my_shmem_size;
  if (hipMemcpyFromSymbol(&my_shmem_size, HIP_SYMBOL(shmem_size), sizeof(my_shmem_size))
      != hipSuccess) {
    return kmcudaMemoryCopyError;
  }
  my_shmem_size *= sizeof(uint32_t);
  if (hipMemsetAsync(ccounts, 0, clusters_size * sizeof(uint32_t)) != hipSuccess) {
    return kmcudaRuntimeError;
  }
  if (hipMemsetAsync(assignments, 0xff, samples_size * sizeof(uint32_t)) != hipSuccess) {
    return kmcudaRuntimeError;
  }
  for (int i = 1; ; i++) {
    kmeans_assign<<<sgrid, sblock>>>(
        samples, centroids, assignments_prev, assignments);
    uint32_t changed_ = 0;
    if (hipMemcpyFromSymbol(&changed_, HIP_SYMBOL(changed), sizeof(changed_))
        != hipSuccess) {
      return kmcudaMemoryCopyError;
    }
    if (verbosity > 0) {
      printf("iteration %d: %" PRIu32 " reassignments\n", i, changed_);
    }
    if (changed_ <= tolerance * samples_size) {
      break;
    }
    changed_ = 0;
    if (hipMemcpyToSymbolAsync(HIP_SYMBOL(changed), &changed_, sizeof(changed_))
        != hipSuccess) {
      return kmcudaMemoryCopyError;
    }
    kmeans_adjust<<<cblock, cgrid, my_shmem_size>>>(
        samples, centroids, assignments_prev, assignments, ccounts);
  }
  return kmcudaSuccess;
}
}
